#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <random>
#include <time.h>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/copy.h>

using std::vector;
using std::random_device;
using std::mt19937;
using std::uniform_real_distribution;

#define SIZE 100000

int main()
{
  clock_t start = clock();

  random_device rnd;
  mt19937 mt;
  mt.seed(rnd());
  uniform_real_distribution<double> real_rnd(0.0, 1.0);

  vector<double> host_w(SIZE, 10.0);
  vector<double> host_input(SIZE);
  vector<double> host_output(SIZE);

  for(int i = 0; i < SIZE; ++i) host_input[i] = real_rnd(mt);

  // allocate device side vector
  thrust::device_vector<double> device_w(SIZE);
  thrust::device_vector<double> device_input(SIZE);
  thrust::device_vector<double> device_output(SIZE);

  // copy host to device
  thrust::copy(host_w.begin(), host_w.end(), device_w.begin());
  thrust::copy(host_input.begin(), host_input.end(), device_input.begin());

  // device_output = device_w * device_input
  clock_t transform_start = clock();
  thrust::transform(device_w.begin(), device_w.end(), device_input.begin(), device_output.begin(), thrust::multiplies<double>());
  clock_t transform_end = clock();

  // copy device to host
  thrust::copy(device_output.begin(), device_output.end(), host_output.begin());

  clock_t end = clock();

  std::cout << "transform time: " << (double)(transform_end - transform_start) / CLOCKS_PER_SEC << std::endl;
  std::cout << "program time: " << (double)(end - start) / CLOCKS_PER_SEC << std::endl;

  return 0;
}
